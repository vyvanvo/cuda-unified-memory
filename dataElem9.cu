/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>

class Managed 
{
public:
  void *operator new(size_t len) {
    void *ptr;
    hipMallocManaged(&ptr, len);
    hipDeviceSynchronize();
    return ptr;
  }

  void operator delete(void *ptr) {
    hipDeviceSynchronize();
    hipFree(ptr);
  }
};

struct DataElement : public Managed
{
  int *num;
  int value;
};

__global__ 
void Kernel(DataElement *elem) {
  printf("On host: num=(%d, %d, %d), value=%d\n", elem->num[0], elem->num[1], elem->num[2], elem->value);

  elem->num[0] = 255;
  elem->value++;
}

void launch(DataElement *elem) {
  Kernel<<< 1, 1 >>>(elem);
  hipDeviceSynchronize();
}

int main(void)
{
  DataElement *e = new DataElement;
  
  e->value = 10;
  hipMallocManaged((void**)&(e->num), sizeof(int) * 3 );
  
  for (int i = 0; i < 3; i++) {
    e->num[i] = 10;
  }

  launch(e);

  printf("On host: num=(%d, %d, %d), value=%d\n", e->num[0], e->num[1], e->num[2], e->value);

  hipFree(e->num);
  delete e;

  hipDeviceReset();
}

