/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>

//POLYMORPHISM

// Managed Base Class -- inherit from this to automatically 
// allocate objects in Unified Memory
class Managed 
{
public:
  void *operator new(size_t len) {
    void *ptr;
    hipMallocManaged(&ptr, len);
    hipDeviceSynchronize();
    return ptr;
  }

  void operator delete(void *ptr) {
    hipDeviceSynchronize();
    hipFree(ptr);
  }
};

// Color Class for Managed Memory
class Color : public Managed
{
  protected:
    int length;
    int *data;

    void _realloc() {
      if (data != 0) {
        hipFree(data);
      }

      length = 3;
      hipMallocManaged(&data, sizeof(int)*length);
    }

  public:
    Color() {
      length = 3;
      //data = new int[3];
      _realloc();
    }
    
    // Constructor for C-string initializer
    Color(const int *s) {
      length = 3;
      //data = new int[3]; 

      _realloc();

      /*for(int i = 0; i < 3; i++) {
        data[i] = s[i];
      }
      */

      memcpy(data, s, sizeof(int)*length);

    }

    // Copy constructor
    Color(const Color& s) {
      length = 3;
      //data = new int[3]; 

      _realloc();

      /*for(int i = 0; i < 3; i++) {
        data[i] = s.data[i];
      }*/

      memcpy(data, s.data, sizeof(int)*3);
      
    }
    
    ~Color() { 
      hipFree(data); 
    }

    // Assignment operator
    Color& operator=(const int* s) {
      length = 3;
      //data = new int[3]; 
      _realloc();

      /*for(int i = 0; i < 3; i++) {
        data[i] = s->data[i];
      }*/

      memcpy(data, s, sizeof(int)*length);

      return *this;
    }

    // Element access (from host or device)
    __host__ __device__
    int& operator[](int pos) { 
      return data[pos];
    }

    // get data
    __host__ __device__
    int* get_color() { return data; }

    // get length
    __host__ __device__
    int get_length() { return length; }

    // virtual function add
    __host__ __device__
    virtual void add() = 0;

};

class Red: public Color, public Managed {
  private:
    int hex = 0xFF0000;

  public:
    __host__ __device__
    void add() { length+=10; }
};

class Yellow: public Color, public Managed {
  private:
    int hex = 0xFFFF00;

  public:
    __host__ __device__
    void add() { length+=20; }
};

class Blue: public Color, public Managed {
  private:
    int hex = 0x0000FF;

  public:
    __host__ __device__
    void add() { length+=30; }
};


struct DataElement : public Managed
{
  Blue color;
  int value;
};


__global__ 
void Kernel_by_pointer(DataElement *elem) {
  //printf("On device by pointer: color=(%d, %d, %d), value=%d, color_length=%d\n", elem->color[0], elem->color[1], elem->color[2], elem->value, elem->color.get_length());

  elem->color[0] = 255;
  elem->value+=10;
  elem->color.add();

  printf("On device by pointer: color=(%d, %d, %d), value=%d, color_length=%d\n", elem->color[0], elem->color[1], elem->color[2], elem->value, elem->color.get_length());
}

__global__ 
void Kernel_by_ref(DataElement &elem) {
  //printf("On device by ref: color=(%d, %d, %d), value=%d, color_length=%d\n", elem.color[0], elem.color[1], elem.color[2], elem.value, elem.color.get_length());

  elem.color[1] = 255;
  elem.value+=20;
  elem.color.add();

  printf("On device by ref: color=(%d, %d, %d), value=%d, color_length=%d\n", elem.color[0], elem.color[1], elem.color[2], elem.value, elem.color.get_length());
}

__global__ 
void Kernel_by_value(DataElement elem) {
  //printf("On device by value: color=(%d, %d, %d), value=%d, color_length=%d\n", elem.color[0], elem.color[1], elem.color[2], elem.value, elem.color.get_length());

  elem.color[2] = 255;
  elem.value+=30;
  elem.color.add();

  printf("On device by ref: color=(%d, %d, %d), value=%d, color_length=%d\n", elem.color[0], elem.color[1], elem.color[2], elem.value, elem.color.get_length());
}

void launch_by_pointer(DataElement *elem) {
  dim3 dim_grid(1, 1, 1);
  dim3 dim_block(1, 1, 1);

  //printf("launch by pointer: name=(%d, %d, %d), value=%d\n", elem->color[0], elem->color[1], elem->color[2], elem->value);
  Kernel_by_pointer<<< dim_grid, dim_block >>>(elem);
  hipDeviceSynchronize();
}

void launch_by_ref(DataElement &elem) {
  dim3 dim_grid(1, 1, 1);
  dim3 dim_block(1, 1, 1);

  //printf("launch by ref: name=(%d, %d, %d), value=%d\n", elem.color[0], elem.color[1], elem.color[2], elem.value);
  Kernel_by_ref<<< dim_grid, dim_block >>>(elem);
  hipDeviceSynchronize();
}

void launch_by_value(DataElement elem) {
  dim3 dim_grid(1, 1, 1);
  dim3 dim_block(1, 1, 1);

  //printf("launch by value: name=(%d, %d, %d), value=%d\n", elem.color[0], elem.color[1], elem.color[2], elem.value);
  Kernel_by_value<<< dim_grid, dim_block >>>(elem);
  hipDeviceSynchronize();
}


int main(void)
{
  DataElement *e = new DataElement;

  
  for (int i = 0; i < 3; i++) {
    e->color[i] = 0;
  }

  e->value = 10;

  printf("On host (print): color=(%d, %d, %d), value=%d, color_length=%d\n", e->color[0], e->color[1], e->color[2], e->value, e->color.get_length());
  //e->color.add();

  printf("On host (after add op): color=(%d, %d, %d), value=%d, color_length=%d\n", e->color[0], e->color[1], e->color[2], e->value, e->color.get_length());

  launch_by_pointer(e);

  printf("On host (after by-pointer): color=(%d, %d, %d), value=%d, , color_length=%d\n", e->color[0], e->color[1], e->color[2], e->value, e->color.get_length());

  launch_by_ref(*e);

  printf("On host (after by-ref): color=(%d, %d, %d), value=%d, , color_length=%d\n", e->color[0], e->color[1], e->color[2], e->value, e->color.get_length());

  launch_by_value(*e);

  printf("On host (after by-value): color=(%d, %d, %d), value=%d, , color_length=%d\n", e->color[0], e->color[1], e->color[2], e->value, e->color.get_length());

  delete e;

  hipDeviceReset();
  
}


